
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <iostream>

#define ARRAY_SIZE 200
#define ARRAY_BYTES ARRAY_SIZE * sizeof(float)

__global__ void CalculateSquare(float* p_out, float* p_in)
{
	int index = threadIdx.x;
	float valueToSuqare = p_in[index];
	p_out[index] = valueToSuqare * valueToSuqare;
}

int main()
{
	float in[ARRAY_SIZE];
	for (int i = 0; i < ARRAY_SIZE; i++)
	{
		in[i] = static_cast<float>(i);
	}
	float out[ARRAY_SIZE];

	float* GPU_in;
	float* GPU_out;

	//Memory allocation in GPU
	hipMalloc((void **)&GPU_in, ARRAY_BYTES);
	hipMalloc((void **)&GPU_out, ARRAY_BYTES);

	//copy(send) result array to GPU
	hipMemcpy(GPU_in, in, ARRAY_BYTES, hipMemcpyHostToDevice);

	//launch the kernal(the __global__ implementation above)
	CalculateSquare<<<1,ARRAY_SIZE>>>(GPU_out, GPU_in);

	//COPY calculated data from GPU to cpu
	hipMemcpy(out, GPU_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

	for (int i = 0; i < ARRAY_SIZE; i++)
	{
		std::cout << out[i] << "     ";
		if (i % 4 == 0)
			std::cout << std::endl;
	}
	//DON'T FORGET TO FREE MEMORY
	hipFree(GPU_in);
	hipFree(GPU_out);
	system("Pause");

	return 0;
}